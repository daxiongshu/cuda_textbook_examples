#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <stdio.h>
#include "common.h"
__global__ void checkIndex(void){
   printf("threadIdx:(%d,%d,%d) blockIdx:(%d,%d,%d) blockDim:(%d,%d,%d) "
        "gridDim: (%d,%d,%d)\n", threadIdx.x,threadIdx.y,threadIdx.z,
        blockIdx.x,blockIdx.y,blockIdx.z,blockDim.x,blockDim.y,
        blockDim.z,gridDim.x,gridDim.y,gridDim.z
   );

}

void checkResult(float *hostRef, float *gpuRef, const int N){
   
   double epsilon= 1.0E-8;
   bool match = 1;
   for(int i=0;i<N;i++){
      if (abs(hostRef[i]-gpuRef[i])>epsilon){
         match=0;
         printf("Arrays don't match!\n");
         printf("host %5.2f gpu %5.2f at current %d\n",hostRef[i],gpuRef[i],i);
         break;
      }
   }
   if (match) printf("Arrays match. \n\n");
}

void sumArraysOnHost(float *A, float *B, float *C, const int N){
   // CPU version of the kernel
   for(int idx=0; idx<N;idx++){
      C[idx]=A[idx]+B[idx];
   }
   // no need to return C
}

void initialData(float *ip, int size){
   time_t t;
   srand((unsigned int) time(&t)); 
   // initialize random number with seed of time
   // time(&t) <==> t=time(NULL); assign the current time to t


   for (int i=0; i<size; i++){
      ip[i] = (float) ( rand() & 0xFF )/10.0f; // rand() returns a random number between 0 and RAND_MAX
   }
}

int main(int argc, char **argv){
   int nElem = 6;//1024;
   size_t nBytes = nElem * sizeof(float);

   /*******************CPU part********************/
   
   float *h_A, *h_B, *h_C;
   h_A = (float *)malloc(nBytes);
   h_B = (float *)malloc(nBytes);
   h_C = (float *)malloc(nBytes);

   initialData(h_A,nElem);
   initialData(h_B,nElem);

   sumArraysOnHost(h_A,h_B,h_C,nElem);

   /*******************CPU part********************/

   


   /*******************GPU part********************/
   dim3 block(3);
   dim3 grid((nElem+block.x-1)/block.x); // how the grid is calculated?

   printf("grid.x %d grid.y %d grid.z %d \n",grid.x,grid.y,grid.z);
   printf("block.x %d block.y %d block.z %d \n",block.x,block.y,block.z);

   // understand index
   checkIndex <<<grid,block>>>();
   hipDeviceReset();

   float *d_A, *d_B, *d_C;
   hipMalloc((float**)&d_A, nBytes);
   hipMalloc((float**)&d_B, nBytes);
   hipMalloc((float**)&d_C, nBytes);

   CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
   CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));

   //hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);




   free(h_A);
   free(h_B);
   free(h_C);

   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);

   return(0);

}
