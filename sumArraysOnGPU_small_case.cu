#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <stdio.h>
#include "common.h"

inline double cpuSecond()
{
    struct timeval tp;
    //struct timezone tzp;
    int i = gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}


__global__ void checkIndex(void){
   printf("threadIdx:(%d,%d,%d) blockIdx:(%d,%d,%d) blockDim:(%d,%d,%d) "
        "gridDim: (%d,%d,%d)\n", threadIdx.x,threadIdx.y,threadIdx.z,
        blockIdx.x,blockIdx.y,blockIdx.z,blockDim.x,blockDim.y,
        blockDim.z,gridDim.x,gridDim.y,gridDim.z
   );

}

__global__ void sumArraysOnGPU(float *A, float *B, float *C)
{  
   // why no boundry check?
   int i=threadIdx.x;
   C[i]=A[i]+B[i];

}
void checkResult(float *hostRef, float *gpuRef, const int N){
   
   double epsilon= 1.0E-8;
   bool match = 1;
   for(int i=0;i<N;i++){
      if (abs(hostRef[i]-gpuRef[i])>epsilon){
         match=0;
         printf("Arrays don't match!\n");
         printf("host %5.2f gpu %5.2f at current %d\n",hostRef[i],gpuRef[i],i);
         break;
      }
   }
   if (match) printf("Arrays match. \n\n");
}

void sumArraysOnHost(float *A, float *B, float *C, const int N){
   // CPU version of the kernel
   for(int idx=0; idx<N;idx++){
      C[idx]=A[idx]+B[idx];
   }
   // no need to return C
}

void initialData(float *ip, int size){
   time_t t;
   srand((unsigned int) time(&t)); 
   // initialize random number with seed of time
   // time(&t) <==> t=time(NULL); assign the current time to t


   for (int i=0; i<size; i++){
      ip[i] = (float) ( rand() & 0xFF )/10.0f; // rand() returns a random number between 0 and RAND_MAX
      //0xFF is 255
   }
}

int main(int argc, char **argv){
   printf("%s Starting ...\n",argv[0]);

   int nElem = 320;//1024;
   printf("Vector size is %d\n",nElem);

   // allocate memory
   size_t nBytes = nElem * sizeof(float);
   //printf("%d,%d\n",0xFF,RAND_MAX);
   /*******************CPU part********************/
   
   float *h_A, *h_B, *h_C, *gpuRef;
   // h_C = h_A + h_B
   // d_C copy to gpuRef
   h_A = (float *)malloc(nBytes);
   h_B = (float *)malloc(nBytes);
   h_C = (float *)malloc(nBytes);
   gpuRef = (float *)malloc(nBytes);

   initialData(h_A,nElem);
   initialData(h_B,nElem);
   memset(h_C,0,nBytes);
   memset(gpuRef,0,nBytes);

   double iStart,iElaps;
   iStart=cpuSecond();
   sumArraysOnHost(h_A,h_B,h_C,nElem);
   iElaps=cpuSecond()-iStart;
   printf("sumArraysOnHost Time Elapsed %f\n",iElaps);
   /*******************CPU part********************/

   


   /*******************GPU part********************/
   int dev=0;
   hipSetDevice(dev); // use the device_id=0 GPU;

   dim3 block(nElem);
   dim3 grid((nElem+block.x-1)/block.x); // how the grid is calculated?

   // understand index
   //checkIndex <<<grid,block>>>();
   //hipDeviceReset();

   float *d_A, *d_B, *d_C;
   hipMalloc((float**)&d_A, nBytes);
   hipMalloc((float**)&d_B, nBytes);
   hipMalloc((float**)&d_C, nBytes);

   CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
   CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));

   iStart=cpuSecond();
   sumArraysOnGPU<<<grid,block>>>(d_A, d_B, d_C);
   CHECK(hipDeviceSynchronize());
   iElaps=cpuSecond()-iStart;
   printf("sumArraysOnGPU Time Elapsed %f\n",iElaps);

   printf("Kernel configuration: (%d,%d,%d),(%d,%d,%d)\n",grid.x,grid.y,grid.z,block.x,block.y,block.z);
   hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

   checkResult(h_C,gpuRef,nElem);





   free(h_A);
   free(h_B);
   free(h_C);
   free(gpuRef);

   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);
   return(0);
}
